#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// CUDA kernel for forward pass
template <typename scalar_t>
__global__ void hgru_real_forward_kernel(
    const scalar_t* x, const scalar_t* lambda,
    scalar_t* output, int64_t n, int64_t b, int64_t d) {
    // Compute the global indices of the current thread
    // batch
    int64_t idy = blockIdx.y * blockDim.y + threadIdx.y;
    // feature
    int64_t idz = blockIdx.x * blockDim.x + threadIdx.x;

    if (idy < b && idz < d) {
        scalar_t hidden_state = 0;
        for (int64_t idx = 0; idx < n; ++idx) {
            int64_t index = idx * b * d + idy * d + idz;
            hidden_state = lambda[index] * hidden_state + x[index];
            output[index] = hidden_state;
        }
    }
}

// CUDA kernel for backward pass
template <typename scalar_t>
__global__ void hgru_real_backward_kernel(
    const scalar_t* x, const scalar_t* lambda, const scalar_t* hidden_states, const scalar_t* grad_output,
    scalar_t* grad_x, scalar_t* grad_lambda,
    int64_t n, int64_t b, int64_t d) {
    // batch
    int64_t idy = blockIdx.y * blockDim.y + threadIdx.y;
    // feature
    int64_t idz = blockIdx.x * blockDim.x + threadIdx.x;

    if (idy < b && idz < d) {
        scalar_t grad_hidden_state = 0;
        for (int64_t idx = n - 1; idx >= 0; --idx) {
            int64_t index = idx * b * d + idy * d + idz;
            int64_t j = ((idx == n - 1) ? 0 : index + b * d);
            grad_hidden_state = grad_output[index] + lambda[j] * grad_hidden_state;

            grad_lambda[index] = grad_hidden_state * ((idx == 0) ? 0 : hidden_states[index - b * d]);
            grad_x[index] = grad_hidden_state;
        }
    }
}

torch::Tensor hgru_real_forward_cuda(
    torch::Tensor &x, torch::Tensor &lambda) {
    auto output = torch::zeros_like(x);
    const int64_t n = x.size(0);
    const int64_t b = x.size(1);
    const int64_t d = x.size(2);

    dim3 threads(128, 8);
    dim3 blocks((d + threads.x - 1) / threads.x, (b + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "hgru_real_forward_cuda", ([&] {
        hgru_real_forward_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(), lambda.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(), n, b, d);
    }));

    return output;
}

std::vector<torch::Tensor> hgru_real_backward_cuda(
    torch::Tensor &x, torch::Tensor &lambda, torch::Tensor &hidden_states, torch::Tensor &grad_output) {
    auto grad_x = torch::zeros_like(x);
    auto grad_lambda = torch::zeros_like(lambda);

    const int64_t n = x.size(0);
    const int64_t b = x.size(1);
    const int64_t d = x.size(2);

    dim3 threads(128, 8);
    dim3 blocks((d + threads.x - 1) / threads.x, (b + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(grad_output.scalar_type(), "hgru_real_backward_cuda", ([&] {
        hgru_real_backward_kernel<scalar_t><<<blocks, threads>>>(
            x.data_ptr<scalar_t>(), lambda.data_ptr<scalar_t>(), hidden_states.data_ptr<scalar_t>(), grad_output.data_ptr<scalar_t>(),
            grad_x.data_ptr<scalar_t>(), grad_lambda.data_ptr<scalar_t>(), n, b, d);
    }));

    return {grad_x, grad_lambda};
}